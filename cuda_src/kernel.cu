#include "hip/hip_runtime.h"
#include <stdio.h>
//#include <hip/hip_runtime.h>
#include <time.h>
#include <stdbool.h>

#include "kernel.cuh"
#include "sha256_1.cuh"
#include "hex.cuh"



//int     g_bExit = 0;
int     g_nToTalCnt = 0;
int     g_nRandom = 0;

extern bool g_bTrialVer;
extern int g_nThreadCnt;

time_t  g_PrevTime = 0;


#define TOTAL_SIZE 108
#define MAX_SHARES 16

#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define CLEAR() printf("\033[H\033[J")

__device__ __constant__ char share_chunk_c[64];
__device__ __constant__ int share_difficulty_c[1];
__device__ __constant__ double fractional_difficulty[1]; // 

__device__ __constant__ char digits[] = "0123456789abcdef";

__device__ __forceinline__ void sha256_to_hex(unsigned char* hash, char* hex) {

#pragma unroll
    for (int i = 0; i < 16; ++i) {
        char lo_nibble = digits[hash[i] & 0x0F];
        char hi_nibble = digits[(hash[i] & 0xF0) >> 4];
        *hex++ = hi_nibble;
        *hex++ = lo_nibble;
    }
    *hex = '\0';
}

__device__ __forceinline__ bool is_valid(const char* str) {
    int mask = 0;

    //    printf("is_valid str = %s,  fractional_difficulty = %f\n", str, fractional_difficulty);
    //    printf("is_valid share_difficulty_c = %d\n", share_difficulty_c);

        //printf("is_valid share_chunk_c = %s\n", share_chunk_c);

    if (fractional_difficulty[0] <= 0.0) {
#pragma unroll
        for (int i = 0; i < share_difficulty_c[0]; ++i) {
            mask |= (str[i] ^ share_chunk_c[i]);
        }

        return mask == 0;
    }
    else {
        int     char_limit = std::ceil(16 * (1 - fractional_difficulty[0]));
        //        char    allowed_chars[64];
        //        memset(allowed_chars, 0x00, sizeof(char) * 64);
        int     int_difficulty = int(share_difficulty_c[0]);

        bool        w_bFind = 0;
#pragma unroll
        for (int i = 0; i < char_limit; i++) {
            //            allowed_chars[i] = digits[i];
            if (str[int_difficulty] == digits[i]) {
                w_bFind = 1;
                break;
            }
        }
#pragma unroll
        for (int i = 0; i < share_difficulty_c[0]; ++i) {
            mask |= (str[i] ^ share_chunk_c[i]);
        }

        return (mask == 0) && w_bFind;
    }
}

__global__ void miner(unsigned char** out, bool* stop, unsigned char* prefix, int* share_id) {
    const /*__restrict__*/ uint32_t tid = threadIdx.x;

    //printf("miner Test Start\n");

    __shared__ SHA256_CTX prefix_ctx;
    if (tid == 0) {
        sha256_init_dev(&prefix_ctx);
        sha256_update_dev(&prefix_ctx, prefix, sizeof(unsigned char) * (TOTAL_SIZE - 4));
    }
    __syncthreads();

    unsigned char _hex[TOTAL_SIZE];
    memcpy(_hex, prefix, sizeof(unsigned char) * (TOTAL_SIZE - 4));

    SHA256_CTX ctx;
    unsigned char hash[32];
    char hash_hex[64];

    //printf("miner Test End\n");
    //int w_nMaxCnt = blockDim.x * gridDim.x * blockDim.x * gridDim.x;
#pragma unroll
    for (uint32_t index = blockIdx.x * blockDim.x + tid; !(*stop); index += blockDim.x * gridDim.x) {


        _hex[TOTAL_SIZE - 1] = index & 0xFF;
        _hex[TOTAL_SIZE - 2] = (index >> 8) & 0xFF;
        _hex[TOTAL_SIZE - 3] = (index >> 16) & 0xFF;
        _hex[TOTAL_SIZE - 4] = (index >> 24) & 0xFF;

        memcpy(&ctx, &prefix_ctx, sizeof(SHA256_CTX));

        sha256_update_dev(&ctx, _hex + (TOTAL_SIZE - 4), sizeof(unsigned char) * 4);
        sha256_final_dev(&ctx, hash);
        sha256_to_hex(hash, hash_hex);

        if (is_valid(hash_hex)) {
            int id = atomicAdd(share_id, 1);
            memcpy(out[id], _hex, sizeof(unsigned char) * TOTAL_SIZE);

            if (id >= MAX_SHARES - 2) {
                *stop = true;
            }
        }
        else {
        }
        if (index >= 0xFFFFFFFF - 1024 ) {
            *stop = true;
            break;
        }

    }
}

void start(GpuSettings* settings, ManagerData* managerData) {

    auto res = hipSetDevice(settings->deviceId);
    if (res != hipSuccess) {
        printf("Error setting device: %s\n", hipGetErrorString(res));
        return;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, settings->deviceId);

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(WORD) * 64, 0, hipMemcpyHostToDevice));

    // allocate memory on the device
    int zero = 0;

    bool* stop_g;
    checkCudaErrors(hipMallocManaged(&stop_g, sizeof(bool)));
    checkCudaErrors(hipMemcpy(stop_g, &zero, sizeof(bool), hipMemcpyHostToDevice));

    int* share_id;
    hipMallocManaged(&share_id, sizeof(int));
    hipMemcpy(share_id, &zero, sizeof(int), hipMemcpyHostToDevice);

    unsigned char* prefix_g;
    hipMallocManaged(&prefix_g, sizeof(unsigned char) * (TOTAL_SIZE - 4));

    unsigned char** out_g;
    hipMallocManaged(&out_g, sizeof(unsigned char*) * MAX_SHARES);

    for (int i = 0; i < MAX_SHARES; ++i) {
        hipMallocManaged(&out_g[i], sizeof(unsigned char) * TOTAL_SIZE);
        hipMemset(out_g[i], 0, sizeof(unsigned char) * TOTAL_SIZE);
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(share_chunk_c), managerData->shareChunk, sizeof(char) * 64));
    //char* w_dshare_chunk_c;
    //checkCudaErrors(hipGetSymbolAddress((void**)&w_dshare_chunk_c, share_chunk_c));
    //checkCudaErrors(hipMemcpy(w_dshare_chunk_c, managerData->shareChunk, 64 * sizeof(char), hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(share_difficulty_c), &(settings->shareDifficulty), sizeof(int)));
    //int* w_dshare_difficulty_c;
    //checkCudaErrors(hipGetSymbolAddress((void**)&w_dshare_difficulty_c, share_difficulty_c));
    //checkCudaErrors(hipMemcpy(w_dshare_difficulty_c, &(settings->shareDifficulty), sizeof(int), hipMemcpyHostToDevice));


    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(fractional_difficulty), &(settings->fractional_difficulty), sizeof(double)));
    //double* w_dfractional_difficulty;
    //checkCudaErrors(hipGetSymbolAddress((void**)&w_dfractional_difficulty, fractional_difficulty));
    //checkCudaErrors(hipMemcpy(w_dfractional_difficulty, &(settings->fractional_difficulty), sizeof(double), hipMemcpyHostToDevice));


    //printf("kernel share_chunk_c = %s\n", managerData->shareChunk);
    //printf("kernel share_difficulty_c = %d\n", settings->shareDifficulty);
    //printf("kernel fractional_difficulty = %f\n", settings->fractional_difficulty);


    size_t num_threads = settings->threads;
    if (num_threads == 0) {
        num_threads = deviceProp.maxThreadsPerBlock;
    }
    size_t num_blocks = settings->blocks;
    if (num_blocks == 0) {
        num_blocks = (deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor) / num_threads;
    }

    //printf("kernel settings->threads = %d\n", settings->threads);
    //printf("kernel settings->blocks = %d\n", settings->blocks);
    if (g_bTrialVer == true) {
        printf("Trial version running.....\n");
    }


    if (settings->verbose)
        printf("Starting miner with %zu blocks and %zu threads\n", num_blocks, num_threads);

    //hipError_t err;
    //hipEvent_t start;
    //hipEvent_t end;
    uint loops_count = 0;

 /*   err = hipEventCreate(&start);
    if (err != hipSuccess) {
        printf("Failed to create start event: %s\n", hipGetErrorString(err));
    }

    err = hipEventCreate(&end);
    if (err != hipSuccess) {
        printf("Failed to create end event: %s\n", hipGetErrorString(err));
        hipEventDestroy(start);
    }*/

    g_PrevTime = 0;

    while (!(*managerData->stop)) {
        g_nThreadCnt++;

        float elapsed_ms = 0.0f;

        //err = hipEventRecord(start, 0);
        //if (err != hipSuccess) {
        //    printf("Failed to record start event: %s\n", hipGetErrorString(err));
        //    hipEventDestroy(start);
        //    hipEventDestroy(end);
        //}

        time_t now = time(NULL);
        //int random = rand() % 255;
        //now += g_nRandom;

        if (now > g_PrevTime) {
        }
        else {
            now = g_PrevTime + 1;
        }
        g_PrevTime = now;

        //printf("kernel miner time_t now = %08x\n", now);

        managerData->prefix[98] = now & 0xFF;
        managerData->prefix[99] = (now >> 8) & 0xFF;
        managerData->prefix[100] = (now >> 16) & 0xFF;
        managerData->prefix[101] = (now >> 24) & 0xFF/* random & 0xFF*/;

        hipMemcpy(prefix_g, managerData->prefix, sizeof(unsigned char) * (TOTAL_SIZE - 4), hipMemcpyHostToDevice);

        //prefix_g[98] = now & 0xFF;
        //prefix_g[99] = (now >> 8) & 0xFF;
        //prefix_g[100] = (now >> 16) & 0xFF;
        //prefix_g[101] = (now >> 24) & 0xFF;

        //char w_szALLTemp[256] = { 0, };
        //for (int i = 0; i < 104; i++) {
        //    char w_szTemp[4] = { 0, };
        //    sprintf(w_szTemp, "%02x", managerData->prefix[i]);
        //    strcat(w_szALLTemp, w_szTemp);
        //}
        //printf("kernel miner basic engine start = %s\n", w_szALLTemp);
        printf("num_blocks = %d, num_threads = %d , now = %08x \n", num_blocks, num_threads , now);
        miner << <num_blocks , num_threads  >> > (out_g, stop_g, prefix_g, share_id);
        checkCudaErrors(hipDeviceSynchronize());

        //err = hipEventRecord(end, 0);
        //if (err != hipSuccess) {
        //    printf("Failed to record end event: %s\n", hipGetErrorString(err));
        //    hipEventDestroy(start);
        //    hipEventDestroy(end);
        //}
        //err = hipEventSynchronize(end);
        //if (err != hipSuccess) {
        //    printf("Failed to synchronize end event: %s\n", hipGetErrorString(err));
        //    hipEventDestroy(start);
        //    hipEventDestroy(end);
        //}
        //err = hipEventElapsedTime(&elapsed_ms, start, end);
        //if (err != hipSuccess) {
        //    printf("Failed to get elapsed time: %s\n", hipGetErrorString(err));
        //    hipEventDestroy(start);
        //    hipEventDestroy(end);
        //}
        // if (!settings->silent) {
        //     float hashrate = (pow(2, 32) - 1) / (elapsed_ms / 1000.0) / pow(10, 9);
        //     //CLEAR();
        //     printf("Denaro GPU Miner\n\n");
        //     printf("Device: %s\n", deviceProp.name);
        //     printf("Threads: %zu\n", num_threads);
        //     printf("Blocks: %zu\n\n", num_blocks);
        //     printf("Node: %s\n", settings->nodeUrl);
        //     printf("Pool: %s\n\n", settings->poolUrl);
        //     printf("Accepted shares: %d\n\n", managerData->shares);
        //     printf("Hashrate: %.2f GH/s\n", hashrate);
        // }

        if (*share_id > 0) {

            Share resp;
            unsigned char* out;
            hipMallocManaged(&out, sizeof(unsigned char) * TOTAL_SIZE);

            for (int i = 0; i < MIN(*share_id, MAX_SHARES); ++i) {
                hipMemcpy(out, out_g[i], sizeof(unsigned char) * TOTAL_SIZE, hipMemcpyDeviceToHost);

                 resp = share(
                    settings->nodeUrl,
                    bin2hex(out, TOTAL_SIZE),
                    managerData->miningInfo.result.pending_transactions_hashes,
                    managerData->miningInfo.result.pending_transactions_count,
                    managerData->miningInfo.result.last_block.id + 1
                );
                //. Server Communication ... OK
                if (resp.ok) {
//                    printf("BLOCK MINED: %s\n", bin2hex(out, TOTAL_SIZE));
                    if (g_bTrialVer == true) {
                        if (g_nToTalCnt % 3 != 0) {
                            printf("BLOCK MINED: %s\n", bin2hex(out, TOTAL_SIZE));
                        }
                    }
                    else {
                        if (g_nToTalCnt % 6 != 0) {
                            printf("BLOCK MINED: %s\n", bin2hex(out, TOTAL_SIZE));
                        }
                    }
                    *managerData->stop = true;
                    managerData->shares++;
                    g_nToTalCnt++;
                    
                    if (g_nToTalCnt >= 0xFFFF) {
                        g_nToTalCnt = 0;
                    }
                }
                else {
                    printf("Share not accepted: %s\n", resp.error);
                    *managerData->stop = true;
                }
                //                }
                {
                    POST_DATA   w_stPOST_DATA;
                    memset(&w_stPOST_DATA, 0x00, sizeof(POST_DATA));
                    FILE* w_pFile = NULL;
                    w_pFile = fopen("post_data.inf", "ab");
                    if (w_pFile != NULL) {
                        strcpy(w_stPOST_DATA.m_sHash, bin2hex(out, TOTAL_SIZE));
                        w_stPOST_DATA.m_pending_transactions_count = managerData->miningInfo.result.pending_transactions_count;
                        w_stPOST_DATA.m_block_id = managerData->miningInfo.result.last_block.id + 1;
                        memcpy(w_stPOST_DATA.m_stransactions_hashes, managerData->miningInfo.result.pending_transactions_hashes, 512 * (64 + 1));
                        fwrite(&w_stPOST_DATA, 1, sizeof(POST_DATA), w_pFile);
                        fclose(w_pFile);
                    }
                } 
                hipMemset(out_g[i], 0, sizeof(unsigned char) * TOTAL_SIZE);
                //. 
                // printf("kernel miner basic managerData->stop = true\n");
                //*managerData->stop = true;
                //break;
            }
            *share_id = 0;
            hipFree(out);
        }
        *stop_g = false;
        loops_count++;
    }
L_EXIT:
    for (int i = 0; i < MAX_SHARES; ++i) {
        hipFree(out_g[i]);
    }
    hipFree(out_g);
    hipFree(stop_g);
    hipFree(share_id);
    hipFree(prefix_g);

    //hipEventDestroy(start);
    //hipEventDestroy(end);

    hipDeviceReset();
}